#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "rgbConversion.h"
#include "Timer.h"
#include "Masks.h"

#define MASK_RADIUS 1
#define MASK_WIDTH 3 
#define MASK_SIZE 9
#define O_TILE_WIDTH 30
#define BLOCK_WIDTH O_TILE_WIDTH + MASK_WIDTH - 1

#define CUDA_CHECK( value ) {									\
	hipError_t err = value;									\
	if( err != hipSuccess ) {									\
		fprintf( stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(err), __LINE__, __FILE__ );	\
		exit( 1 );												\
	} 															\
}


__global__ void convolution(const uchar* N, const float* __restrict__ M, uchar* P, int height, int width) {

	// shared memory 
	__shared__ uchar Ns[BLOCK_WIDTH][BLOCK_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row_o = blockIdx.y * O_TILE_WIDTH + ty;
	int col_o = blockIdx.x * O_TILE_WIDTH + tx;

	int row_i = row_o - MASK_RADIUS;
	int col_i = col_o - MASK_RADIUS;

	// copying data to shared memory
	if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width)) {
		Ns[ty][tx] = N[row_i * width + col_i];
	}
	else {
		Ns[ty][tx] = 0.0f;
	}

	// wait until data are copied in all threads
	__syncthreads();

	// convolution calculation
	float result = 0.0f;

	if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {
		for (int i = 0; i < MASK_WIDTH; ++i) {
			for (int j = 0; j < MASK_WIDTH; ++j) {
				result += M[i * MASK_WIDTH + j] * Ns[i + ty][j + tx];
			}
		}

		if (row_o < height && col_o < width) {
			P[row_o * width + col_o] = static_cast<uchar>(result);
		}
	}

	__syncthreads();
}



int main()
{
	// input image
	png_img_t inputImage;
	std::string imageInputName = "images/6k_png_img.png";
	std::cout << "image: " << imageInputName << '\n';
	inputImage.read(imageInputName);
	int imageWidth = inputImage.get_width();
	int imageHeight = inputImage.get_height();
	int imageSize = imageWidth * imageHeight;

	// *******************************************************************
	//                    memory allocation on CPU
	// *******************************************************************
	uchar* host_input_red = (uchar*)malloc(sizeof(uchar) * imageSize);
	uchar* host_input_green = (uchar*)malloc(sizeof(uchar) * imageSize);
	uchar* host_input_blue = (uchar*)malloc(sizeof(uchar) * imageSize);
	uchar* host_output_red = (uchar*)malloc(sizeof(uchar) * imageSize);
	uchar* host_output_green = (uchar*)malloc(sizeof(uchar) * imageSize);
	uchar* host_output_blue = (uchar*)malloc(sizeof(uchar) * imageSize);

	// *******************************************************************
	//                    memory allocation on GPU
	// *******************************************************************
	uchar* device_input_red = NULL, * device_input_green = NULL, * device_input_blue = NULL;
	CUDA_CHECK(hipMalloc((void**)&device_input_red, sizeof(uchar) * imageSize));
	CUDA_CHECK(hipMalloc((void**)&device_input_green, sizeof(uchar) * imageSize));
	CUDA_CHECK(hipMalloc((void**)&device_input_blue, sizeof(uchar) * imageSize));

	uchar* device_output_red = NULL, * device_output_green = NULL, * device_output_blue = NULL;
	CUDA_CHECK(hipMalloc((void**)&device_output_red, sizeof(uchar) * imageSize));
	CUDA_CHECK(hipMalloc((void**)&device_output_green, sizeof(uchar) * imageSize));
	CUDA_CHECK(hipMalloc((void**)&device_output_blue, sizeof(uchar) * imageSize));

	// *******************************************************************
	//         get RGB input data and copy them from CPU to GPU
	// *******************************************************************
	pngToRgb(inputImage, host_input_red, host_input_green, host_input_blue);

	CUDA_CHECK(hipMemcpy(device_input_red, host_input_red, sizeof(uchar) * imageSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_input_green, host_input_green, sizeof(uchar) * imageSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(device_input_blue, host_input_blue, sizeof(uchar) * imageSize, hipMemcpyHostToDevice));

	// *******************************************************************
	//   memory allocation of mask on CPU, pick data and copy to GPU
	// *******************************************************************
	float* host_mask = (float*)malloc(sizeof(float) * MASK_SIZE);
	Masks maskMatrices;
	std::string strOutputName = "boxBlur";
	std::cout << "convolution matrix effect: " << strOutputName << '\n';
	for (int i = 0; i < MASK_SIZE; ++i) {
		host_mask[i] = maskMatrices.boxBlur[i];
	}

	float* device_mask = NULL;
	CUDA_CHECK(hipMalloc((void**)&device_mask, sizeof(float) * MASK_SIZE));
	CUDA_CHECK(hipMemcpy(device_mask, host_mask, sizeof(float) * MASK_SIZE, hipMemcpyHostToDevice));

	// *******************************************************************
	//                          calculation
	// *******************************************************************
	// number of threads and block size
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid((imageWidth + O_TILE_WIDTH - 1) / O_TILE_WIDTH, (imageHeight + O_TILE_WIDTH - 1) / O_TILE_WIDTH);

	Timer timer;
	timer.start();

	convolution <<<dimGrid, dimBlock>>> (device_input_red, device_mask, device_output_red, imageHeight, imageWidth);
	convolution <<<dimGrid, dimBlock>>> (device_input_green, device_mask, device_output_green, imageHeight, imageWidth);
	convolution <<<dimGrid, dimBlock>>> (device_input_blue, device_mask, device_output_blue, imageHeight, imageWidth);

	timer.stop("time of all rgb data calculation");

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipGetLastError());

	// *******************************************************************
	//        copy calculated values from GPU to CPU and save them
	// *******************************************************************
	CUDA_CHECK(hipMemcpy(host_output_red, device_output_red, sizeof(uchar) * imageSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(host_output_green, device_output_green, sizeof(uchar) * imageSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(host_output_blue, device_output_blue, sizeof(uchar) * imageSize, hipMemcpyDeviceToHost));

	png_img_t outputImage(imageWidth, imageHeight);
	rgbToPng(host_output_red, host_output_green, host_output_blue, outputImage);
	std::string outputPath = "images/output_6k_";
	outputPath.append(strOutputName);
	outputPath.append(".png");
	outputImage.write(outputPath);

	// *******************************************************************
	//            free all allocated GPU and CPU memory
	// *******************************************************************

	CUDA_CHECK(hipFree((void*)device_input_red));
	CUDA_CHECK(hipFree((void*)device_input_green));
	CUDA_CHECK(hipFree((void*)device_input_blue));
	CUDA_CHECK(hipFree((void*)device_output_red));
	CUDA_CHECK(hipFree((void*)device_output_green));
	CUDA_CHECK(hipFree((void*)device_output_blue));

	free(host_input_red);
	free(host_input_green);
	free(host_input_blue);
	free(host_output_red);
	free(host_output_green);
	free(host_output_blue);

	CUDA_CHECK(hipDeviceReset());

    return 0;
}


